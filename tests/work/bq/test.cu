#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <hiprand/hiprand.h>
// #include "helper.cuh"

typedef half    Real;
typedef half2   Complex;
__global__ void a()
{
  __half a ;
  a = __hsub(__half(1),__half(2));
}
int main(int argc, char *argv[])
{

    long long sample_size = 1<<20;      // 1,048,576
    const int batch_size = 1<<9;//1 << 9;  // 512

    hipfftHandle plan_forward, plan_inverse;
    Real    *p_sample;
    Complex *d_freq;

    float forward_time_ms, inverse_time_ms;

    // create cuda event to measure the performance
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // create host buffer as input data

    hipMallocManaged((void**)&p_sample, sizeof(Real) * sample_size * batch_size);

    a<<<1,1>>>();
    for (int i=0;i<sample_size;i++)
    {
        float a= 12.4322;
        p_sample[i] = uint(half(a));
        printf("%d", p_sample[i]);
    }
    return 0;
    // create signal and filter memory
    hipMallocManaged((void**)&d_freq,   sample_size * sizeof(Complex) * batch_size);

    // 1D cufft setup
    int rank = 1;
    int stride_sample = 1, stride_freq = 1;
    long long int dist_sample = sample_size, dist_freq = sample_size / 2 + 1;
    long long embed_sample[] = {0};
    long long embed_freq[] = {0};
    size_t workSize = 0;
    hipfftCreate(&plan_forward);
    hipfftXtMakePlanMany(plan_forward, 
        rank, &sample_size, 
        embed_sample, stride_sample, dist_sample, HIP_R_16F, 
        embed_freq, stride_freq, dist_freq, HIP_C_16F, 
        batch_size, &workSize, HIP_C_16F);
    hipfftCreate(&plan_inverse);
    hipfftXtMakePlanMany(plan_inverse,
    	rank, &sample_size,
        embed_freq, stride_freq, dist_freq, HIP_C_16F,
        embed_sample, stride_sample, dist_sample, HIP_R_16F,
        batch_size, &workSize, HIP_R_16F);

    // executes FFT processes
    hipEventRecord(start);
    hipfftXtExec(plan_forward, p_sample, d_freq, HIPFFT_FORWARD);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&forward_time_ms, start, stop);

    // for (int i=0;i<sample_size/2+1;i++)
    // {
    //     printf("%f %f \n", (d_freq[i].x),(d_freq[i].y));// = __float2half(1);
    // } 

    // executes FFT processes (inverse transformation)
    hipEventRecord(start);
    hipfftXtExec(plan_inverse, d_freq, p_sample, HIPFFT_BACKWARD);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&inverse_time_ms, start, stop);
    // for (int i=0;i<sample_size;i++)
    // {
    //     printf("%f \n", (p_sample[i]));// = __float2half(1);
    // } 

    
    // print elapsed time
    //printf("FFT operation time for %d" << sample_size << " elements with " << batch_size << " batch.." << std::endl;
    printf("Forward (ms): %f\n" ,forward_time_ms);// << std::endl;
    printf("Inverse (ms): %f\n" ,inverse_time_ms);// << std::endl;

    // deletes CUFFT plan_forward
    hipfftDestroy(plan_forward);
    hipfftDestroy(plan_inverse);

    // terminates memories
    hipFree(p_sample);
    
    // delete cuda events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}